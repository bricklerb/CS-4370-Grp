#include <stdio.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#define BLOCK_SIZE 2
#define ARRAY_SIZE 8

void prefix_sum(int *arrayB, int *arrayA, int array_size);
__global__ void prefix_sum_kernel(int *dev_arrayA, int *arrayB, int array_size);
void initArray(int *, bool, int);
void displayArray(int *, int);

int main()
{
    // save ARRAY_SIZE to local just in case
    int inputSize = ARRAY_SIZE;
    // CPU define arrays
    int arrayA[ARRAY_SIZE];
    int arrayB[ARRAY_SIZE];
    // device arrays
    int *dev_arrayA;
    int *dev_arrayB; // TODO: don't need dev_arrayB?
    int *dev_output;
    hipMalloc((void **)&dev_arrayA, (inputSize * sizeof(int)));
    hipMalloc((void **)&dev_arrayB, (inputSize * sizeof(int)));
    hipMalloc((void **)&dev_output, (inputSize * sizeof(int)));

    // thread block size
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(ceil(double(ARRAY_SIZE)/(2 * dimBlock.x)));

    // init arrays
    initArray(arrayA, true, ARRAY_SIZE);
    initArray(arrayB, true, ARRAY_SIZE);

    // copy content from cpu arrayA/B to gpu dev_arrayA/B
    hipMemcpy(dev_arrayA, arrayA, (inputSize * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dev_arrayB, arrayB, (inputSize * sizeof(int)), hipMemcpyHostToDevice); // TODO: don't need dev_arrayB?

    // seg fault ?
    int *dev_outputOnHost = new int[inputSize];
    hipMemcpy(dev_output, dev_outputOnHost, (inputSize * sizeof(int)), hipMemcpyHostToDevice);

    std::cout << "Starting array: " << std::endl; 
    displayArray(arrayA, ARRAY_SIZE);
    std::cout << std::endl;

    // CPU prefix sum
    prefix_sum(arrayB, arrayA, ARRAY_SIZE);
    std::cout << "CPU array: " << std::endl;
    displayArray(arrayB, ARRAY_SIZE);
    std::cout << std::endl;

    

    // GPU prefix sum
    prefix_sum_kernel<<<dimGrid,dimBlock>>>(dev_arrayA, dev_arrayB, ARRAY_SIZE); // TODO: don't need dev_arrayB?

    // seg fault
    hipMemcpy(dev_outputOnHost, dev_arrayA, (inputSize * sizeof(int)), hipMemcpyDeviceToHost);

    std::cout << "GPU parallel prefix sum" << std:: endl;
    displayArray(dev_outputOnHost, ARRAY_SIZE);
    std::cout << std::endl;
}

void prefix_sum(int *arrayB, int *arrayA, int array_size)
{
    // this is sequential sum
    arrayB[0] = arrayA[0];
    for (int i = 1; i < array_size; i++)
        arrayB[i] = arrayB[i - 1] + arrayA[i];
}

// TODO: don't need dev_arrayB?
__global__ void prefix_sum_kernel(int *dev_arrayA, int *dev_arrayB, int array_size)
{

    __shared__ int scan_array[2 * BLOCK_SIZE];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;
    scan_array[t] = dev_arrayA[start + t];
    scan_array[blockDim.x + t] = dev_arrayA[start + blockDim.x + t];

    __syncthreads();

    //  Reduction step
    int stride = 1;
    int index;

    while (stride <= BLOCK_SIZE)
    {
        index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index < 2 * BLOCK_SIZE)
            scan_array[index] += scan_array[index - stride];
        stride = stride * 2;

        __syncthreads();
    }

    // Post Scan Step
    stride = BLOCK_SIZE / 2;
    while (stride > 0)
    {
        index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < 2 * BLOCK_SIZE)
        {
            scan_array[index + stride] += scan_array[index];
        }
        stride = stride / 2;
        __syncthreads();
    }

    __syncthreads();

    dev_arrayA[start + t] = scan_array[t];
    dev_arrayA[start + blockDim.x + t] = scan_array[blockDim.x + t];
}

/// @brief Displays the contents of a given array as a matrix, if the MATRIX_WIDTH is larger than 8 -> only displays first row
/// @param matrix The given array for the matrix
void displayArray(int *array, int arraySize)
{
    std::cout << "[ ";

    if (arraySize > 19)
    {
        arraySize = 20;
    }

    for (int i = 0; i < arraySize; i++)
    {
        std::cout << array[i] << " ";
    }

    std::cout << "]" << std::endl;
}

/// @brief Sets all the values in an array to zero for initialization
/// @param array
void initArray(int *array, bool initData, int arraySize)
{
    // Loop through rows and columns and init values to 0
    for (int i = 0; i < arraySize; i++)
    {
        array[i] = 0;
    }

    if (initData)
    {
        int init = 1325;
        for (int i = 0; i < arraySize; i++)
        {
            init = 3125 * init % 6553;
            array[i] = (init - 1000) % 97;
        }
    }
}
