#include <stdio.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024
#define ARRAY_SIZE 2048

void prefix_sum(int *arrayB, int *arrayA, int array_size);
__global__ void prefix_sum_kernel(int *dev_arrayA, int *arrayB, int array_size);
void initArray(int *, bool, int);
void displayArray(int *, int);
bool arraysEqual(int*, int*);

int main()
{
    //-------------//
    //CPU EXECUTION//
    //-------------//


    // save ARRAY_SIZE to local just in case
    int inputSize = ARRAY_SIZE;
    // CPU define arrays
    int arrayA[ARRAY_SIZE];
    int arrayB[ARRAY_SIZE];

    // init arrays
    initArray(arrayA, true, ARRAY_SIZE);
    initArray(arrayB, true, ARRAY_SIZE);

    std::cout << "Starting array: " << std::endl; 
    displayArray(arrayA, ARRAY_SIZE);
    std::cout << std::endl;

    // measure cpu runtime
    clock_t start, end;
    start = clock();

    // CPU prefix sum
    prefix_sum(arrayB, arrayA, ARRAY_SIZE);
    end = clock(); // end cpu runtime

    // Display how long it took the CPU to execute
    printf("\nCLOCKS_PER_SEC:%ld", CLOCKS_PER_SEC);
    printf("\nNumber of clock ticks:%ld", (end - start));
    printf("\nCPU execution time in seconds:%f\n", (double)(end - start) / CLOCKS_PER_SEC);

    std::cout << "CPU array: " << std::endl;
    displayArray(arrayB, ARRAY_SIZE);
    std::cout << std::endl;

    //-------------//
    //GPU EXECUTION//
    //-------------//

    // device arrays
    int *dev_arrayA;
    int *dev_arrayB; // TODO: don't need dev_arrayB?
    int *dev_output;
    hipMalloc((void **)&dev_arrayA, (inputSize * sizeof(int)));
    hipMalloc((void **)&dev_arrayB, (inputSize * sizeof(int)));
    hipMalloc((void **)&dev_output, (inputSize * sizeof(int)));

    // thread block size
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(ceil(double(ARRAY_SIZE)/(2 * dimBlock.x)));


    // copy content from cpu arrayA/B to gpu dev_arrayA/B
    hipMemcpy(dev_arrayA, arrayA, (inputSize * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dev_arrayB, arrayB, (inputSize * sizeof(int)), hipMemcpyHostToDevice); // TODO: don't need dev_arrayB?

    // seg fault ?
    int *dev_outputOnHost = new int[inputSize];
    hipMemcpy(dev_output, dev_outputOnHost, (inputSize * sizeof(int)), hipMemcpyHostToDevice);



    std::cout << "Number of thread blocks: " << dimGrid.x << std::endl;
    std::cout << "Thread block size: " << dimBlock.x << std::endl;

    // loop for gpu run time
    float timeGPU;
    hipEvent_t gpuStart, gpuStop;

    hipEventCreate(&gpuStart);
    hipEventCreate(&gpuStop);
    hipEventRecord(gpuStart, 0);

    // GPU prefix sum
    // prefix_sum_kernel<<<dimGrid,dimBlock>>>(dev_arrayA, dev_arrayB, ARRAY_SIZE); // TODO: don't need dev_arrayB?
    prefix_sum_kernel<<<8192,1024>>>(dev_arrayA, dev_arrayB, ARRAY_SIZE); // TODO: don't need dev_arrayB?

    hipDeviceSynchronize();
    hipEventRecord(gpuStop, 0);
    hipEventSynchronize(gpuStop);
    hipEventElapsedTime(&timeGPU, gpuStart, gpuStop);
    hipEventDestroy(gpuStart);
    hipEventDestroy(gpuStop);


    // seg fault
    hipMemcpy(dev_outputOnHost, dev_arrayA, (inputSize * sizeof(int)), hipMemcpyDeviceToHost);

    // Display results
    // std::cout << "GPU Execution Time in seconds: " << totalGPUTime / 60 << std::endl;
    printf("\nGPU execution time in seconds:%f\n", (timeGPU));

    std::cout << "GPU parallel prefix sum" << std:: endl;
    displayArray(dev_outputOnHost, ARRAY_SIZE);
    std::cout << std::endl;

    if (arraysEqual(dev_outputOnHost, arrayB)) printf("TEST PASSED!\n");
    else (printf("TEST FAILED!\n"));

}

void prefix_sum(int *arrayB, int *arrayA, int array_size)
{
    // this is sequential sum
    arrayB[0] = arrayA[0];
    for (int i = 1; i < array_size; i++)
        arrayB[i] = arrayB[i - 1] + arrayA[i];
}

// TODO: don't need dev_arrayB? 
// not needed but i dont want to break anything
__global__ void prefix_sum_kernel(int *dev_arrayA, int *dev_arrayB, int array_size)
{
    __shared__ int scan_array[2 * BLOCK_SIZE];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;
    scan_array[t] = dev_arrayA[start + t];
    scan_array[blockDim.x + t] = dev_arrayA[start + blockDim.x + t];

    __syncthreads();

    //  Reduction step
    int stride = 1;
    int index;

    while (stride <= BLOCK_SIZE)
    {
        index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index < 2 * BLOCK_SIZE)
            scan_array[index] += scan_array[index - stride];
        stride = stride * 2;

        __syncthreads();
    }

    // Post Scan Step
    stride = BLOCK_SIZE / 2;
    while (stride > 0)
    {
        index = (threadIdx.x + 1) * stride * 2 - 1;

        if ((index + stride) < 2 * BLOCK_SIZE)
        {
            scan_array[index + stride] += scan_array[index];
        }
        stride = stride / 2;
        __syncthreads();
    }

    __syncthreads();

    dev_arrayA[start + t] = scan_array[t];
    dev_arrayA[start + blockDim.x + t] = scan_array[blockDim.x + t];
}

/// @brief Displays the contents of a given array as a matrix, if the MATRIX_WIDTH is larger than 8 -> only displays first row
/// @param matrix The given array for the matrix
void displayArray(int *array, int arraySize)
{
    std::cout << "[ ";

    if (arraySize > 19)
    {
        arraySize = 20;
    }

    for (int i = 0; i < arraySize; i++)
    {
        std::cout << array[i] << " ";
    }

    std::cout << "]" << std::endl;
}

/// @brief Sets all the values in an array to zero for initialization
/// @param array
void initArray(int *array, bool initData, int arraySize)
{
    // Loop through rows and columns and init values to 0
    for (int i = 0; i < arraySize; i++)
    {
        array[i] = 0;
    }

    if (initData)
    {
        int init = 1325;
        for (int i = 0; i < arraySize; i++)
        {
            init = 3125 * init % 6553;
            array[i] = (init - 1000) % 97;
        }
    }
}

bool arraysEqual(int* a, int* b)
{
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        if (a[i] != b[i]) return false;
    }
    return true;
}
