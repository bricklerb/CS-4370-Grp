// Students: Jahcorian Ivery, Bella Brickler, Renee Paxson
// Class: CS 4370 - Parallel Programming for Many-core Gpus
// Instructor: Meilin Liu
// Date - 12/03/24
// Assignment: Project 4

#include <stdio.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024
#define ARRAY_SIZE 4096

void histogram(unsigned int *buffer, unsigned int *histo);
void initArray(unsigned int *, bool, int);
void displayArray(unsigned int *matrix, int);
void displayHistogram(unsigned int *hist);
__global__ void histo_kernel(unsigned int *buffer, long size, unsigned int *histo);
bool arrayEqual(unsigned int *array1, unsigned int *array2, long size);

int main()
{
    // Define the matricies necessary for matrix addition
    unsigned array[ARRAY_SIZE];
    unsigned int cpuHisto[256];
    unsigned int deviceHistoOnHost[256];
    // unsigned int histo_private[256];

    // Init  each arry with data
    initArray(array, true, ARRAY_SIZE);
    initArray(cpuHisto, false, 256);

    std::cout << "Array size: " << ARRAY_SIZE << std::endl;

    clock_t start, end; // used to measure the execution time on CPU
    start = clock();

    // Do Sum reduction on CPU
    histogram(array, cpuHisto);

    end = clock();

    // Display how long it took the CPU to execute
    printf("\nCLOCKS_PER_SEC:%ld", CLOCKS_PER_SEC);
    printf("\nNumber of clock ticks:%ld", (end - start));
    printf("\nCPU execution time in seconds:%f\n", (double)(end - start) / CLOCKS_PER_SEC);

    std::cout << "Array:" << std::endl;
    displayArray(array, ARRAY_SIZE);
    std::cout << std::endl;

    std::cout << "CPU Output:" << std::endl;
    displayHistogram(cpuHisto);
    std::cout << std::endl;

    // Declare the array on the device
    unsigned int *deviceArray;
    unsigned int *deviceHisto;

    hipMalloc((void **)&deviceArray, (ARRAY_SIZE * sizeof(unsigned int)));
    hipMalloc((void **)&deviceHisto, (256 * sizeof(unsigned int)));

    // Reset data in histogram
    initArray(deviceHistoOnHost, false, 256);

    hipMemcpy(deviceArray, array, ARRAY_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(deviceHisto, deviceHistoOnHost, ARRAY_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

    int dimBlock = BLOCK_SIZE;
    int dimGrid = 1;

    std::cout << "Number of thread blocks: " << dimGrid << std::endl;
    std::cout << "Thread block size: " << dimBlock << std::endl;

    
    float timeGPU; // Time the GPU method.
    hipEvent_t gpuStart, gpuStop;

    hipEventCreate(&gpuStart);
    hipEventCreate(&gpuStop);
    hipEventRecord(gpuStart, 0);

    //     parallel_sum_reduction<<<dimGrid, dimBlock>>>(gpuInput);
    histo_kernel<<<dimGrid, dimBlock>>>(deviceArray, ARRAY_SIZE, deviceHisto);

    hipDeviceSynchronize();

    hipEventRecord(gpuStop, 0);
    hipEventSynchronize(gpuStop);
    hipEventElapsedTime(&timeGPU, gpuStart, gpuStop);
    hipEventDestroy(gpuStart);
    hipEventDestroy(gpuStop);

    hipMemcpy(deviceHistoOnHost, deviceHisto, (256 * sizeof(unsigned int)), hipMemcpyDeviceToHost);

    // Display results
    std::cout << "GPU Execution Time in seconds: " << timeGPU / 60 << std::endl;

    std::cout << "GPU Output:" << std::endl;
    displayHistogram(deviceHistoOnHost);
    std::cout << std::endl;

    hipFree(deviceHisto);
    hipFree(deviceArray);

    if (arrayEqual(deviceHistoOnHost, cpuHisto, 256))
    {
        std::cout << "TEST PASSED" << std::endl;
    }
}

/// @brief Displays the first 10 elements of an array
/// @param matrix The given array
void displayHistogram(unsigned int *hist)
{
    std::cout << "[ ";

    for (int i = 0; i < 10; i++)
    {
        std::cout << hist[i] << " ";
    }

    std::cout << "]" << std::endl;
}

/// @brief Displays the first 10 elements of an array
/// @param matrix The given array
void displayArray(unsigned int *array, int arraySize)
{
    std::cout << "[ ";

    if (arraySize > 10)
    {
        arraySize = 10;
    }

    for (int i = 0; i < arraySize; i++)
    {
        std::cout << array[i] << " ";
    }

    std::cout << "]" << std::endl;
}

/// @brief Sets all the values in an array to zero for initialization
/// @param array
void initArray(unsigned int *array, bool initData, int arraySize)
{
    // Loop through rows and columns and init values to 0
    for (int i = 0; i < arraySize; i++)
    {
        array[i] = 0;
    }

    if (initData)
    {
        int init = 1325;
        for (int i = 0; i < arraySize; i++)
        {
            init = 3125 * init % 65537;
            array[i] = init % 256;
        }
    }
}

void histogram(unsigned int *buffer, unsigned int *histo)
{
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        histo[buffer[i]] += 1;
    }
}

__global__ void histo_kernel(unsigned int *buffer, long size, unsigned int *histo)
{
    __shared__ unsigned int histo_private[256];
    if (threadIdx.x < 256) histo_private[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // stride is total number of threads
    int stride = blockDim.x * gridDim.x; // All threads handle blockDim.x * gridDim.x
    // consecutive elements in one loop iteration
    while (i < size)
    {
        atomicAdd(&(histo_private[buffer[i]]), 1);
        i += stride;
    }

    // printf("%d", histo[i]);
    
    if (threadIdx.x < 256) 
    {
        atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);
    }
    
}

bool arrayEqual(unsigned int *array1, unsigned int *array2, long size)
{
    for (long i = 0; i < size; i++)
    {
        if (array1[i] != array2[i])
        {
            return false;
        }
    }

    return true;
}
