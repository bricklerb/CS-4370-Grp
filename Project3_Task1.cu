// Students: Jahcorian Ivery, Bella Brickler, Renee Paxson
// Class: CS 4370 - Parallel Programming for Many-core Gpus
// Instructor: Meilin Liu
// Date - 11/8/2024
// Assignment: Project 3

#include <stdio.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

// #define MATRIX_WIDTH 4096
#define BLOCK_SIZE 4
#define ARRAY_SIZE 16
// #define TILE_WIDTH 32

int sum_reduction(int *x, int N);
void initArray(int *, bool, int);
void displayArray(int *matrix, int);
__global__ void parallel_sum_reduction(int *input);

int main()
{
    // Define the matricies necessary for matrix addition
    int array[ARRAY_SIZE];
    int outputCpuArray[ARRAY_SIZE];

    // Init  each arry with data
    initArray(array, true, ARRAY_SIZE);
    initArray(outputCpuArray, true, ARRAY_SIZE);

    std::cout << "Array size: " << ARRAY_SIZE << std::endl;

    clock_t start, end; // used to measure the execution time on CPU
    start = clock();

    // Do Sum reduction on CPU
    sum_reduction(outputCpuArray, ARRAY_SIZE);

    end = clock();

    // Display how long it took the CPU to execute
    printf("\nCLOCKS_PER_SEC:%ld", CLOCKS_PER_SEC);
    printf("\nNumber of clock ticks:%ld", (end - start));
    printf("\nCPU execution time in seconds:%f\n", (double)(end - start) / CLOCKS_PER_SEC);

    std::cout << "Array:" << std::endl;
    // displayArray(array, ARRAY_SIZE);
    std::cout << std::endl;

    std::cout << "CPU Output:" << std::endl;
    // displayArray(outputCpuArray, ARRAY_SIZE);
    std::cout << std::endl;

    // Declare the array on the device
    int inputSize = ARRAY_SIZE;
    int *gpuInput;

    hipMalloc((void **)&gpuInput, (inputSize * sizeof(int)));
    hipMemcpy(gpuInput, array, inputSize * sizeof(int), hipMemcpyHostToDevice);

    int dimBlock = BLOCK_SIZE;
    int dimGrid = ceil((double)ARRAY_SIZE / (2 * dimBlock));

    // Loop!!
    while (true)
    {
        std::cout << "Thread block size: " << dimBlock << std ::endl;
        std::cout << "Number of thread blocks: " << dimGrid << std::endl;

        // // // Create timing variables
        // float timeGPU; // Time the GPU method.
        // cudaEvent_t gpuStart, gpuStop;

        // cudaEventCreate(&gpuStart);
        // cudaEventCreate(&gpuStop);
        // cudaEventRecord(gpuStart, 0);

        // parallel_sum_reduction<<<dimGrid, dimBlock>>>(gpuInput);

        // cudaDeviceSynchronize();
        // cudaEventRecord(gpuStop, 0);
        // cudaEventSynchronize(gpuStop);
        // cudaEventElapsedTime(&timeGPU, gpuStart, gpuStop);
        // cudaEventDestroy(gpuStart);
        // cudaEventDestroy(gpuStop);

        // If only one block was used no need to reduce more
        if (dimGrid == 1)
        {
            break;
        }

        // we can reduce in one more execution
        int oldDimGrid = dimGrid;
        if (oldDimGrid < 513)
        {
            dimBlock = ceil(oldDimGrid / 2);
        }

        int dimGrid = ceil((double)oldDimGrid / (2 * dimBlock));
    }

    // int currentOutputSize = dimBlock.x; // Number of thread blocks that will generate answers
    // int *gpuOutputOnHost = new int[currentOutputSize];
    // initArray(gpuOutputOnHost, false);

    // int inputSize = ARRAY_SIZE;

    // float totalGPUTime = 0;

    // // Loop through until only 1 thread block executes to get the final answer
    // // Create arrays on device for this execution
    // int *gpuInput;

    // cudaMalloc((void **)&gpuOutput, (currentOutputSize * sizeof(int)));
    // cudaMalloc((void **)&gpuInput, (inputSize * sizeof(int)));

    // // Copy over the current inputs and
    // cudaMemcpy(gpuInput, array, inputSize * sizeof(int), cudaMemcpyHostToDevice);
    // cudaMemcpy(gpuOutput, gpuOutputOnHost, inputSize * sizeof(int), cudaMemcpyHostToDevice);

    // cudaDeviceSynchronize();
    // cudaEventRecord(gpuStop, 0);
    // cudaEventSynchronize(gpuStop);
    // cudaEventElapsedTime(&timeGPU, gpuStart, gpuStop);
    // cudaEventDestroy(gpuStart);
    // cudaEventDestroy(gpuStop);

    // totalGPUTime += timeGPU;

    // cudaMemcpy(gpuOutputOnHost, gpuOutput, currentOutputSize * sizeof(int), cudaMemcpyDeviceToHost);
    // displayArray(gpuOutputOnHost);

    // // Clean up GPU resources from first reduction
    // cudaFree(gpuInput);
    // cudaFree(gpuOutput);

    // // Reduce the answers if needed
    // inputSize = dimBlock.x;

    // // Input to the next reduction is set to the output of the last run
    // gpuInput = gpuOutputOnHost;
    // inputSize = dimBlock.x; // new input is previous blocks

    // // Change block sizes
    // dimBlock = (inputSize / 2); // half the number of blocks
    // dimGrid = (double)inputSize / (2 * dimBlock.x);

    // // update new output size
    // currentOutputSize = dimBlock.x;

    // cudaDeviceSynchronize();

    // // Display results
    // std::cout << "GPU Execution Time in seconds: " << totalGPUTime << std::endl;

    // std::cout << "GPU Output:" << std::endl;
    // displayArray(gpuOutputOnHost);
    // std::cout << std::endl;
}

/// @brief Displays the contents of a given array as a matrix, if the MATRIX_WIDTH is larger than 8 -> only displays first row
/// @param matrix The given array for the matrix
void displayArray(int *array, int arraySize)
{
    std::cout << "[ ";

    for (int i = 0; i < arraySize; i++)
    {
        std::cout << array[i] << " ";
    }

    std::cout << "]" << std::endl;
}

/// @brief Sets all the values in an array to zero for initialization
/// @param array
void initArray(int *array, bool initData, int arraySize)
{
    // Loop through rows and columns and init values to 0
    for (int i = 0; i < arraySize; i++)
    {
        array[i] = 0;
    }

    if (initData)
    {
        int init = 1325;
        for (int i = 0; i < arraySize; i++)
        {
            init = 3125 * init % 6553;
            array[i] = (init - 1000) % 97;
        }
    }
}

/// @brief Determines if two given matricies are equal to one another
/// @param matrixA
/// @param matrixB
/// @return
bool matrixEqual(float *matrixA, float *matrixB)
{
    // // Loop through rows and columns of matrix
    // for (int i = 0; i < MATRIX_WIDTH; i++)
    // {
    //     for (int j = 0; j < MATRIX_WIDTH; j++)
    //     {

    //         // If a single entry doesnt equal return false
    //         int index = i * MATRIX_WIDTH + j;
    //         if (matrixA[index] != matrixB[index])
    //         {
    //             return false;
    //         }
    //     }
    // }

    // // If we never hit false then the matricies are equal
    return true;
}

int sum_reduction(int *x, int N)
{
    for (int i = 1; i < N; i++)
        x[0] = x[0] + x[i];
    int overallSum = x[0];
    return overallSum;
}

__global__ void parallel_sum_reduction(int *input)
{
    __shared__ int partialSum[2 * BLOCK_SIZE];

    unsigned int tx = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    partialSum[tx] = input[start + tx];
    partialSum[blockDim.x + tx] = input[start + blockDim.x + tx];

    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
    {
        __syncthreads();
        if (tx < stride)
        {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    __syncthreads();
    input[tx + (blockIdx.x * blockDim.x)];
    input[blockIdx.x] = partialSum[0];
}
