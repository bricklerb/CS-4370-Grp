// Students: Jahcorian Ivery, Bella Brickler, Renee Paxson
// Class: CS 4370 - Parallel Programming for Many-core Gpus
// Instructor: Meilin Liu
// Date - 11/8/2024
// Assignment: Project 3

#include <stdio.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

// #define MATRIX_WIDTH 4096
#define BLOCK_SIZE 2
#define ARRAY_SIZE 8
// #define TILE_WIDTH 32

int sum_reduction(int *x, int N);
void initArray(int *array);
void displayArray(int *matrix);
__global__ void parallel_sum_reduction(int *input, int *output);

int main()
{
    // Define the matricies necessary for matrix addition
    int array[ARRAY_SIZE];
    int outputCpuArray[ARRAY_SIZE];

    // Init  each arry with data
    initArray(array);
    initArray(outputCpuArray);

    // Thread block size
    dim3 dimBlock(BLOCK_SIZE);

    dim3 dimGrid = (double)ARRAY_SIZE / (2 * dimBlock.x);
    std::cout << dimGrid.x;

    // Print array information
    std::cout << "Array size: " << ARRAY_SIZE << std::endl;
    std::cout << "Thread block size: " << BLOCK_SIZE << std ::endl;
    std::cout << "Number of thread blocks: " << dimGrid.x << std::endl;

    clock_t start, end; // used to measure the execution time on CPU
    start = clock();

    // Do Sum reduction on CPU
    sum_reduction(outputCpuArray, ARRAY_SIZE);

    end = clock();

    // Display how long it took the CPU to execute
    printf("\nCLOCKS_PER_SEC:%ld", CLOCKS_PER_SEC);
    printf("\nNumber of clock ticks:%ld", (end - start));
    printf("\nCPU execution time in seconds:%f\n", (double)(end - start) / CLOCKS_PER_SEC);

    std::cout << "Array:" << std::endl;
    displayArray(array);
    std::cout << std::endl;

    std::cout << "CPU Output:" << std::endl;
    displayArray(outputCpuArray);
    std::cout << std::endl;

    // // Define GPU arrays
    int *gpuInput, *gpuOutput;

    int currentOutputSize = dimGrid.x; // Number of thread blocks that will generate answers
    int gpuOutputOnHost[currentOutputSize];

    int inputSize = ARRAY_SIZE;

    float totalGPUTime = 0;

    // Loop through until only 1 thread block executes to get the final answer
    // while (currentOutputSize != 1)
    // {
    // Create arrays on device for this execution
    hipMalloc((void **)&gpuInput, inputSize * sizeof(int));
    hipMalloc((void **)&gpuOutput, currentOutputSize * sizeof(int));

    std::cout << currentOutputSize;

    // Copy over the current inputs and
    hipMemcpy(gpuInput, array, inputSize * sizeof(int), hipMemcpyHostToDevice);

    // // Create timing variables
    float timeGPU; // Time the GPU method.
    hipEvent_t gpuStart, gpuStop;

    hipEventCreate(&gpuStart);
    hipEventCreate(&gpuStop);
    hipEventRecord(gpuStart, 0);

    parallel_sum_reduction<<<dimGrid, dimBlock>>>(gpuInput, gpuOutput);

    hipDeviceSynchronize();
    hipEventRecord(gpuStop, 0);
    hipEventSynchronize(gpuStop);
    hipEventElapsedTime(&timeGPU, gpuStart, gpuStop);
    hipEventDestroy(gpuStart);
    hipEventDestroy(gpuStop);

    totalGPUTime += timeGPU;

    hipMemcpy(gpuOutputOnHost, gpuOutput, currentOutputSize * sizeof(int), hipMemcpyDeviceToHost);

    // Input to the next reduction is set to the output of the last run
    gpuInput = gpuOutputOnHost;
    inputSize = sizeof(gpuOutputOnHost);

    // Change block sizes
    dimBlock = (inputSize / 2);
    dimGrid = (double)inputSize / (2 * dimBlock.x);

    // update new output size
    currentOutputSize = dimGrid.x;

    // Clean up GPU resources
    hipFree(gpuInput);
    hipFree(gpuOutput);

    hipDeviceSynchronize();
    // }

    // Display results
    std::cout << "GPU Execution Time in seconds: " << totalGPUTime << std::endl;

    std::cout << "GPU Output:" << std::endl;
    displayArray(gpuOutputOnHost);
    std::cout << std::endl;
}

/// @brief Displays the contents of a given array as a matrix, if the MATRIX_WIDTH is larger than 8 -> only displays first row
/// @param matrix The given array for the matrix
void displayArray(int *array)
{
    std::cout << "[ ";

    for (int i = 0; i < sizeof(array); i++)
    {
        std::cout << array[i] << " ";
    }

    std::cout << "]" << std::endl;

    // if (MATRIX_WIDTH <= 8)
    // {
    //     // Small matrix print entire thing
    //     for (int i = 0; i < MATRIX_WIDTH; i++)
    //     {
    //         // Loop through columns of row
    //         for (int j = 0; j < MATRIX_WIDTH; j++)
    //         {
    //             // Display the value of the current entry
    //             int index = i * MATRIX_WIDTH + j;
    //             if (matrix[index] < 0)
    //             {
    //                 std::cout << matrix[index] << "  ";
    //             }
    //             else
    //             {
    //                 std::cout << matrix[index] << "   ";
    //             }
    //         }
    //         std::cout << std::endl;
    //     }
    // }
    // else
    // {
    //     // large matrix only print first row
    //     for (int i = 0; i < MATRIX_WIDTH; i++)
    //     {
    //         // Display the value of the current entry
    //         std::cout << matrix[i] << " ";
    //     }

    //     std::cout << std::endl;
    // }
}

/// @brief Sets all the values in an array to zero for initialization
/// @param array
void initArray(int *array)
{
    // Loop through rows and columns and init values to 0
    for (int i = 0; i < sizeof(array); i++)
    {
        array[i] = (int)0;
    }

    int init = 1325;
    for (int i = 0; i < sizeof(array); i++)
    {
        init = 3125 * init % 6553;
        array[i] = (init - 1000) % 97;
    }
}

/// @brief Determines if two given matricies are equal to one another
/// @param matrixA
/// @param matrixB
/// @return
bool matrixEqual(float *matrixA, float *matrixB)
{
    // // Loop through rows and columns of matrix
    // for (int i = 0; i < MATRIX_WIDTH; i++)
    // {
    //     for (int j = 0; j < MATRIX_WIDTH; j++)
    //     {

    //         // If a single entry doesnt equal return false
    //         int index = i * MATRIX_WIDTH + j;
    //         if (matrixA[index] != matrixB[index])
    //         {
    //             return false;
    //         }
    //     }
    // }

    // // If we never hit false then the matricies are equal
    return true;
}

int sum_reduction(int *x, int N)
{
    for (int i = 1; i < N; i++)
        x[0] = x[0] + x[i];
    int overallSum = x[0];
    return overallSum;
}

__global__ void parallel_sum_reduction(int *input, int *output)
{
    printf("%d", sizeof(input));

    __shared__ int partialSum[2 * BLOCK_SIZE];

    unsigned int tx = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    partialSum[tx] = input[start + tx];
    partialSum[blockDim.x + tx] = input[start + blockDim.x + tx];

    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
    {
        __syncthreads();
        if (tx < stride)
        {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    __syncthreads();
    output[blockIdx.x] = partialSum[0];
}
